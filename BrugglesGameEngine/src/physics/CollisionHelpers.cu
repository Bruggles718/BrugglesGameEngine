#include "hip/hip_runtime.h"
#include "physics/CollisionHelpers.cuh"
#include "physics/CollisionPoints.cuh"
#include "physics/CollisionObject.cuh"
#include <iostream>
#include "TDynamicArray.cuh"
#include "Transform.cuh"
#include "physics/CircleCollider.cuh"
#include "physics/Simplex.cuh"
#include "physics/PhysicsHelpers.cuh"

namespace bruggles {
	namespace physics {
		__global__ void ComputeCollision(Transform* i_at, Transform* i_bt, Collider** i_ac, Collider** i_bc, bool* i_hasCollision, Simplex* i_simplexes) {
			int i = threadIdx.x;

            auto result = GJK(i_bc[i], &i_bt[i], i_ac[i], &i_at[i]);

            i_hasCollision[i] = result.first;
            i_simplexes[i] = result.second;
		}

		void GPUComputeCollisions(std::vector<std::pair<CollisionObject*, CollisionObject*>>& pairs, std::vector<CollisionPoints>& i_result) {
            if (pairs.size() < 1) {
                return;
            }

            hipDeviceSetLimit(hipLimitMallocHeapSize, 100000);

            // Extract pointers from the vector of pairs
            i_result.resize(pairs.size());
            std::vector<Transform> firstList{};
            std::vector<Transform> secondList{};
            firstList.reserve(pairs.size());
            secondList.reserve(pairs.size());

            for (int i = 0; i < pairs.size(); i++) {
                firstList.push_back(pairs[i].first->GetTransform());
                secondList.push_back(pairs[i].second->GetTransform());
            }

            // Allocate device memory and copy data
            Transform* d_firstList = 0;
            Transform* d_secondList = 0;
            CollisionPoints* d_result = 0;

            Collider** d_firstColliderList = 0;
            Collider** d_secondColliderList = 0;

            float* d_radii = 0;

            hipError_t e;

            e = hipMalloc(&d_firstList, pairs.size() * sizeof(Transform));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMalloc(&d_secondList, pairs.size() * sizeof(Transform));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMalloc(&d_firstColliderList, pairs.size() * sizeof(Collider*));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMalloc(&d_secondColliderList, pairs.size() * sizeof(Collider*));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            std::vector<Collider*> firstColliderList;
            std::vector<Collider*> secondColliderList;
            firstColliderList.reserve(pairs.size());
            secondColliderList.reserve(pairs.size());

            for (int i = 0; i < pairs.size(); i++) {
                Collider* a = pairs[i].first->collider->GetDeviceCopy();
                Collider* b = pairs[i].second->collider->GetDeviceCopy();
                firstColliderList.push_back(a);
                secondColliderList.push_back(b);
            }
            e = hipGetLastError();
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            e = hipMemcpy(d_firstColliderList, firstColliderList.data(), firstColliderList.size() * sizeof(Collider*), hipMemcpyHostToDevice);
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMemcpy(d_secondColliderList, secondColliderList.data(), secondColliderList.size() * sizeof(Collider*), hipMemcpyHostToDevice);
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            e = hipMalloc((void**)&d_result, pairs.size() * sizeof(CollisionPoints));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMalloc((void**)&d_radii, pairs.size() * sizeof(float));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMemcpy(d_firstList, firstList.data(), firstList.size() * sizeof(Transform), hipMemcpyHostToDevice);
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }
            e = hipMemcpy(d_secondList, secondList.data(), secondList.size() * sizeof(Transform), hipMemcpyHostToDevice);
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            bool* d_hasCollision = 0;
            e = hipMalloc(&d_hasCollision, pairs.size() * sizeof(bool));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            Simplex* d_simplexes = 0;
            e = hipMalloc(&d_simplexes, pairs.size() * sizeof(Simplex));
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }

            // Launch kernel
            int numCollisions = pairs.size();
            ComputeCollision << <1, numCollisions >> > (d_firstList, d_secondList, d_firstColliderList, d_secondColliderList, d_hasCollision, d_simplexes);
            hipDeviceSynchronize();
            /*e = hipGetLastError();
            if (e != hipSuccess) {
                std::cout << hipGetErrorString(e) << std::endl;
            }*/
            // Copy result back to host
            /*std::vector<float> i_radii{};
            i_radii.resize(pairs.size());*/
            bool* i_hasCollision = new bool[pairs.size()];
            //hipMemcpy(i_result.data(), d_result, i_result.size() * sizeof(CollisionPoints), hipMemcpyDeviceToHost);
            Vector2* simplex = new Vector2[3];
            hipMemcpy(i_hasCollision, d_hasCollision, pairs.size() * sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(simplex, &d_simplexes[0].Vertices, sizeof(Vector2) * 3, hipMemcpyDeviceToHost);

            //std::cout << std::boolalpha;

            for (int i = 0; i < i_result.size(); i++) {
                std::cout << i << " result: \nHasCollision: " << i_hasCollision[i] << std::endl;
                if (i_hasCollision) {
                    for (int i = 0; i < 3; i++) {
                        std::cout << simplex[i].x << ", " << simplex[i].y << std::endl;
                    }
                }
            }

            // Free device memory
            hipFree(d_firstList);
            hipFree(d_secondList);
            hipFree(d_result);
            hipFree(d_firstColliderList);
            hipFree(d_secondColliderList);
            hipFree(d_hasCollision);
            hipFree(d_simplexes);
            delete[] i_hasCollision;
            delete[] simplex;
		}
	}
}